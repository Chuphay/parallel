#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void use_shared(float *array){

  int i, index = threadIdx.x;
  float average, sum = 0;

  __shared__ float sh_arr[128];

  sh_arr[index]  = array[index];

  __syncthreads();

  for(i = 0; i<=index; i++) sum += sh_arr[i];

  average = sum/(index + 1);

  array[index] = average;

}

int main(){

  float *h_array, *d_array;
  int n = 10;
  int size = n*sizeof(float);
  h_array = (float *) malloc(size);
  hipMalloc((void **) &d_array, size);

  int i;
  for(i = 0; i< n ;i++) h_array[i] = i+1;
  for(i = 0; i< n ; i++) printf("%f\n",h_array[i]);

  hipMemcpy(d_array, h_array, size, hipMemcpyHostToDevice);
//  GpuTimer timer;
 // timer.Start();
  use_shared<<<1,n>>>(d_array);
 // timer.Stop();

  hipMemcpy(h_array, d_array, size, hipMemcpyDeviceToHost);

  for(i = 0; i<n ; i++) printf("%f\n",h_array[i]);
 // printf("Elapsed: %f\n", timer.Elapsed());
  return 0;

}

