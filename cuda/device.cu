
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__device__ int get_global_index(){
return blockIdx.x * blockDim.x + threadIdx.x;
}
__device__ int get_constant(){
return 7;
}
__global__ void kernel1(int *array){
int index = get_global_index();
array[index] = get_constant();
}
__global__ void kernel2(int *array){
int index = get_global_index();
array[index] = get_global_index();
}

int main(){
int num_elements = 256;
int num_bytes = num_elements*sizeof(int);

int *device_array = 0;
int *host_array = 0;

host_array = (int *) malloc(num_bytes);
hipMalloc((void**)&device_array, num_bytes);

int block_size = 128;
int grid_size = num_elements/block_size;

kernel1<<<grid_size, block_size>>>(device_array);
hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

printf("kernel 1 results: \n");
int i;
for(i=0;i<num_elements;i++){
printf("%d ", host_array[i]);
}
printf("\n");

kernel2<<<grid_size, block_size>>>(device_array);
hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

printf("kernel 2 results: \n");
for(i= 0; i< num_elements; i++){
printf("%d ", host_array[i]);
}
printf("\n");
return 0;
}