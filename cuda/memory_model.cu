
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(void){
int num_elements = 16;
int num_bytes = num_elements*sizeof(int);

int *device_array = 0;
int *host_array = 0;

host_array = (int *)malloc(num_bytes);

hipMalloc((void**)&device_array, num_bytes);

hipMemset(device_array, 0, num_bytes);

hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

int i;
for(i  = 0; i<num_elements; i++)
printf("%d\n", host_array[i]);

free(host_array);

hipFree(device_array);

return 0;
}