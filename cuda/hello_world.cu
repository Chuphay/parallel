#include "hip/hip_runtime.h"
#include "util/cuPrintf.cu"
#include <stdio.h>

__global__ void device_greetings(void){
cuPrintf("Hello world from the device!\n");
}

int main(void){
printf("Hello from the host!\n");

cudaPrintfInit();

device_greetings<<<2,3>>>();

cudaPrintfDisplay();

cudaPrintfEnd();

return 0;

}