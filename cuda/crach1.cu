#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void foo(int *ptr){
*ptr = 7;
}

int main(){
foo<<<1,1>>>(0);

hipDeviceSynchronize();

hipError_t error = hipGetLastError();
if(error != hipSuccess){
printf("Cuda error: %s\n", hipGetErrorString(error));
exit(-1);
}

return 0;
}