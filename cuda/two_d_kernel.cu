
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void kernel(int *array){

int index_x = blockIdx.x * blockDim.x + threadIdx.x;
int index_y = blockIdx.y * blockDim.y + threadIdx.y;

int grid_width = gridDim.x * blockDim.x;
int index = index_y * grid_width + index_x; 

int result = blockIdx.y * gridDim.x + blockIdx.x;

array[index] = result;
}

int main(){

int num_elements_x = 16;
int num_elements_y = 16;

int num_bytes = num_elements_x * num_elements_y * sizeof(int);

int *device_array = NULL;
int *host_array = 0;

host_array = (int *) malloc(num_bytes);
hipMalloc((void **) &device_array, num_bytes);

dim3 block_size;

block_size.x = 4;
block_size.y = 4;

dim3 grid_size;
grid_size.x = num_elements_x / block_size.x;
grid_size.y = num_elements_y / block_size.y;

kernel<<<grid_size, block_size>>>(device_array);

hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

int row, col;
for(row = 0; row <num_elements_y; ++row){
for(col = 0; col<num_elements_x; ++col)
printf("%2d ", host_array[row*num_elements_x + col]);

printf("\n");
}

free(host_array);
hipFree(device_array);

return 0;
}