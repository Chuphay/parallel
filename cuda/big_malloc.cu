
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

int main(){
int *ptr = 0;
hipError_t error = hipMalloc((void**)&ptr, UINT_MAX);
if(error != hipSuccess){
printf("CUDA error: %s\n", hipGetErrorString(error));
exit(-1);
}
return 0;
}