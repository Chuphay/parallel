#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_MASSES 1024

__global__ void make_move(float *y, float *yold, float *v, float *out, int time){

  out[time] = yold[NUM_MASSES/2];

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float Ktension = 0.2;
  float Kdamping = 0.9999;

  if ( i > 0 && i < NUM_MASSES-1 ) {
    float accel = Ktension * (yold[i+1] + yold[i-1] - 2*yold[i]);
    //      add accel to velocity
    v[i] += accel;
    v[i] *= Kdamping;
    //      add velocity to position
    y[i] = yold[i] + v[i];
  }
}

void serial(float *y, float *yold, float *v, int numIters){
  float Ktension = 0.2;
  float Kdamping = 0.9999;


  for ( int t = 0; t < numIters; t++ ) {
    // for each mass element
    float sum = 0;
    for ( int i = 0; i < NUM_MASSES; i++ ) {
      //   if boundary element
      //      handle boundary element
      if ( i == 0 || i == NUM_MASSES-1 ) {
      } else {
	//   else
	//      compute acceleration as scaled sum of differences with neighbors
	float accel = Ktension * (yold[i+1] + yold[i-1] - 2*yold[i]);
	//      add accel to velocity
	v[i] += accel;
	v[i] *= Kdamping;
	//      add velocity to position
	y[i] = yold[i] + v[i];
	sum += y[i];
      }
    }
    //   swap displacement buffers
    float *tmp = y;
    y = yold;
    yold = tmp;
  }
}


int main(){

  float *y, *device_y, *yold, *device_yold;
  float *v, *device_v; 
  float *cuda_out;
  
  const float Ktension = 0.2;
  const float Kdamping = 0.9999;
  const float duration = 1.0;

  int size = NUM_MASSES*sizeof(float);
  y = (float *)malloc(size);
  yold = (float *)malloc(size);
  v = (float *)malloc(size);
  hipMalloc((void **) &device_y, size);
  hipMalloc((void **) &device_yold, size);
  hipMalloc((void **) &device_v, size);


  // initialize displacements (pluck it!) and velocities
  for (int i = 0; i < NUM_MASSES; i++ ) {
    v[i]  = 0.0f;
    yold[i] = y[i] = 0.0f;
    if (i == NUM_MASSES/2 )
      yold[i] = 1.0; // impulse at string center
  }

  hipMemcpy(device_y, y, size, hipMemcpyHostToDevice);
  hipMemcpy(device_yold, yold, size, hipMemcpyHostToDevice);
  hipMemcpy(device_v, v, size, hipMemcpyHostToDevice);

  // run simulation for desired period:
  int numIters = duration *20; // 44100; // 5 seconds
  hipMalloc((void **) &cuda_out, numIters*sizeof(float));

  int num_blocks = 8;
  for(int i = 0; i< numIters/2; i++){
    make_move<<<num_blocks, NUM_MASSES/num_blocks>>>(device_y, device_yold, device_v, cuda_out, 2*i);
    make_move<<<num_blocks, NUM_MASSES/num_blocks>>>(device_yold, device_y, device_v, cuda_out, 2*i+1);
  }
  hipDeviceSynchronize();

  float *host_out = (float *)malloc(numIters*sizeof(float));
  hipMemcpy(host_out, cuda_out, numIters*sizeof(float), hipMemcpyDeviceToHost);

  serial(y, yold, v,numIters);
  for(int i = 0; i<numIters; i++)
    printf("%f\n", y[NUM_MASSES/2]);



  return 0;
}
