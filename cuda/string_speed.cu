#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define NUM_MASSES 1024

__global__ void make_move(float *y, float *yold, float *v, float *out, int time){

  out[time] = yold[NUM_MASSES/2];

  int i = blockIdx.x*blockDim.x + threadIdx.x;
  float Ktension = 0.2;
  float Kdamping = 0.9999;

  if ( i > 0 && i < NUM_MASSES-1 ) {
    float accel = Ktension * (yold[i+1] + yold[i-1] - 2*yold[i]);
    //      add accel to velocity
    v[i] += accel;
    v[i] *= Kdamping;
    //      add velocity to position
    y[i] = yold[i] + v[i];
  }
}

void serial(float *y, float *yold, float *v, int numIters, float *out){
  float Ktension = 0.2;
  float Kdamping = 0.9999;


  for ( int t = 0; t < numIters; t++ ) {
    // for each mass element
    float sum = 0;
    for ( int i = 0; i < NUM_MASSES; i++ ) {
      //   if boundary element
      //      handle boundary element
      if ( i == 0 || i == NUM_MASSES-1 ) {
      } else {
	//   else
	//      compute acceleration as scaled sum of differences with neighbors
	float accel = Ktension * (yold[i+1] + yold[i-1] - 2*yold[i]);
	//      add accel to velocity
	v[i] += accel;
	v[i] *= Kdamping;
	//      add velocity to position
	y[i] = yold[i] + v[i];
	sum += y[i];
      }
    }
    //   swap displacement buffers
    float *tmp = y;
    y = yold;
    yold = tmp;
    out[t] = y[NUM_MASSES/2];
  }
}


int main(int argc, char **argv){
  if(argc != 3){
    printf("you got to give me either 1 for cuda or 0 for serial and then a time\n");
    exit(1);
  }
  int cuda = atoi(argv[1]);
  
  float *y, *device_y, *yold, *device_yold;
  float *v, *device_v, *out, *host_out; 
  float *cuda_out;
  
  //const float Ktension = 0.2;
  //const float Kdamping = 0.9999;
  int duration = atoi(argv[2]);

  int size = NUM_MASSES*sizeof(float);
  y = (float *)malloc(size);
  yold = (float *)malloc(size);
  v = (float *)malloc(size);

 

  // initialize displacements (pluck it!) and velocities
  for (int i = 0; i < NUM_MASSES; i++ ) {
    v[i]  = 0.0f;
    yold[i] = y[i] = 0.0f;
    if (i == NUM_MASSES/2 )
      yold[i] = 1.0; // impulse at string center
  }

  // run simulation for desired period:
  int numIters = duration *44100; // 5 seconds
  

  if(cuda){
    hipMalloc((void **) &device_y, size);
    hipMalloc((void **) &device_yold, size);
    hipMalloc((void **) &device_v, size);
    hipMemcpy(device_y, y, size, hipMemcpyHostToDevice);
    hipMemcpy(device_yold, yold, size, hipMemcpyHostToDevice);
    hipMemcpy(device_v, v, size, hipMemcpyHostToDevice);
  
    hipMalloc((void **) &cuda_out, numIters*sizeof(float));

    int num_blocks = 8;
    for(int i = 0; i< numIters/2; i++){
      make_move<<<num_blocks, NUM_MASSES/num_blocks>>>(device_y, device_yold, device_v, cuda_out, 2*i);
      make_move<<<num_blocks, NUM_MASSES/num_blocks>>>(device_yold, device_y, device_v, cuda_out, 2*i+1);
    }
    hipDeviceSynchronize();

    host_out = (float *)malloc(numIters*sizeof(float));
    hipMemcpy(host_out, cuda_out, numIters*sizeof(float), hipMemcpyDeviceToHost);
  } else {

    out = (float *)malloc(numIters*sizeof(float)); 
    serial(y, yold, v,numIters, out);

  }
  
  for(int i = 0; i<20; i++){
    if(cuda)
      printf("%f\n", host_out[i]);
    else
      printf("%f\n", out[i]);
  }

  
  return 0;
}
