#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//uses shared memory, in a single block
//limited to n<= 4000 if only 16K shared memory

__device__ void initsp( int *sprimes, int n, int nth, int me){

int chunk, startsetsp, endsetsp, val, i;

sprimes[2] = 1;

chunk = (n-1)/ nth;
startsetsp = 2 +me*chunk;

if(me<nth -1) endsetsp = startsetsp + chunk-1;
else endstep = n;

val = startsetsp%2;
for(i = startsetsp; i<=endsetsp; i++){
sprimes[i] = val;
val = 1 - val;
}

--syncthreads();
}

__device__ void cpytoglb(int *dprimes, int *sprimes, int n, int nth, int me){
int startcpy, endcpy, chunk, i;
chunk = (n-1)/nth;
startcpy = 2 + me*chunk;
if(me<nth-1) endcpy = startcpy + chunk -1;
else endcpy = n;

for(i = startcpy; i<= endcpy; i++) dprimes[i] = sprimes[i];

__syncthreads();
}

__global__ void sieve(int *dprimes, int n, int nth){

extern __shared__ int sprimes[];
int me = threadIdx.x;
int nthl = nth -1;

initsp(sprimes, n ,nth, me);

int maxmult, m, startmult, endmult, chunk, i;
for(m = 3; m*m <= n; m++){
if(sprimes != 0){
maxmult = n/m;
chunk = (maxmult -1)/nth;
startmult = 2 + me*chunk;
if(me<nthl) endmult = startmult + chunk -1;
else endmult = maxmult;
}
for(i = startmult; i <= endmult; i++) sprimes[i*m] = 0;
}
__syncthreads();
cpytoglb(dprimes,sprimes,n,nth,me);
}

int main(){
int n = 100, nth = 256;

int *hprimes, *dprimes;
int psize = (n+1)*sizeof(int);

hprimes = (int *) malloc(psize);

dim3 dimGrid(1,1);
dim3 dimBlock(nth, 1, 1);

sieve<<<dimGrid, dimBlock, psize>>>(dprimes, n ,nth);
