#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>


//uses shared memory, in a single block
//limited to n<= 4000 if only 16K shared memory

__device__ void initsp( int *sprimes, int n, int nth, int me){

int chunk, startsetsp, endsetsp, val, i;

sprimes[2] = 1;

chunk = (n-1)/ nth;
startsetsp = 2 +me*chunk;

if(me<nth -1) endsetsp = startsetsp + chunk-1;
else endsetsp = n;

val = startsetsp%2;
for(i = startsetsp; i<=endsetsp; i++){
sprimes[i] = val;
val = 1 - val;
}

__syncthreads();
}

__device__ void cpytoglb(int *dprimes, int *sprimes, int n, int nth, int me){
int startcpy, endcpy, chunk, i;
chunk = (n-1)/nth;
startcpy = 2 + me*chunk;
if(me<nth-1) endcpy = startcpy + chunk -1;
else endcpy = n;

for(i = startcpy; i<= endcpy; i++) dprimes[i] = sprimes[i];

__syncthreads();
}

__global__ void sieve(int *dprimes, int n, int nth){

extern __shared__ int sprimes[];
int me = threadIdx.x;
int nthl = nth -1;

initsp(sprimes, n ,nth, me);

int maxmult, m, startmult, endmult, chunk, i;
for(m = 3; m*m <= n; m++){
if(sprimes != 0){
maxmult = n/m;
chunk = (maxmult -1)/nth;
startmult = 2 + me*chunk;
if(me<nthl) endmult = startmult + chunk -1;
else endmult = maxmult;
}
for(i = startmult; i <= endmult; i++) sprimes[i*m] = 0;
}
__syncthreads();
cpytoglb(dprimes,sprimes,n,nth,me);
}

int main(){
printf("start\n");
int n = 10, nth = 16;

int *hprimes, *dprimes;
int psize = (n+1)*sizeof(int);

hprimes = (int *) malloc(psize);
printf("before cuda Malloc\n");

hipMalloc((void **) &dprimes, psize);
printf("after cudamalloc\n");

dim3 dimGrid(1,1);
dim3 dimBlock(nth, 1, 1);

sieve<<<dimGrid, dimBlock, psize>>>(dprimes, n ,nth);

hipError_t err = hipGetLastError();
if(err != hipSuccess) printf("fail\n");
printf("apparently enough memory\n");

hipDeviceSynchronize();

hipMemcpy(hprimes, dprimes, psize, hipMemcpyDeviceToHost);


int i;
for(i = 2; i<=n; i++){
      if(hprimes[i] == 1){ 
      		    printf("%d\n", i);
		    }
}

return 0;
}