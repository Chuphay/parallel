
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void vector_add(const float *a, const float *b, float *c, const size_t n){
unsigned int i = threadIdx.x + blockDim.x*blockIdx.x;

if(i<n)
c[i] = a[i] + b[i];
}

int main(){

const int num_elements = 1<<20;

const int num_bytes = num_elements*sizeof(float);

  float *device_array_a = 0;
  float *device_array_b = 0;
  float *device_array_c = 0;
  float *host_array_a   = 0;
  float *host_array_b   = 0;
  float *host_array_c   = 0;

  // malloc the host arrays
  host_array_a = (float*)malloc(num_bytes);
  host_array_b = (float*)malloc(num_bytes);
host_array_c = (float*)malloc(num_bytes);
if((host_array_c == NULL)||(host_array_b == NULL)||(host_array_a == NULL)){
printf("couldnt allocate that much memory\n");
exit(1);
}

hipMalloc((void**)&device_array_a, num_bytes);
hipMalloc((void**)&device_array_b, num_bytes);
hipMalloc((void**)&device_array_c, num_bytes);



if((device_array_c == NULL)||(device_array_b == NULL)||(device_array_a == NULL)){
printf("couldnt allocate that much memory for the GPU\n");
exit(1);
}

int i;
for(i = 0; i< num_elements; i++){
host_array_a[i]= (float)i;

host_array_b[i] = rand()/((float)RAND_MAX+1.0);
}

hipMemcpy(device_array_a, host_array_a, num_bytes, hipMemcpyHostToDevice);
hipMemcpy(device_array_b, host_array_b, num_bytes, hipMemcpyHostToDevice);

const size_t block_size = 256;
size_t grid_size = num_elements/block_size;

if(num_elements%block_size) ++grid_size;

vector_add<<<grid_size, block_size>>>(device_array_a, device_array_b, device_array_c, num_elements);

hipMemcpy(host_array_c, device_array_c, num_bytes, hipMemcpyDeviceToHost);

for(i = 0; i< 10; i++)
printf("result %d: %1.1f + %7.1f = %7.1f\n",i,host_array_a[i], host_array_b[i], host_array_c[i]);

free(host_array_a);
free(host_array_b);
free(host_array_c);

hipFree(device_array_a);
hipFree(device_array_b);
hipFree(device_array_c);



return 0;
}