#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

//nvcc -o mutual_outlinks mutual_outlinks.cu -arch sm_20



//find mean number of mutual outlinks
//among all pairs of websites
//checking all (i,j) pairs
//thread k will handle all i such that
//i%totth = k, where totth is the number of threads

__global__ void procpairs(int *m, int *tot, int n){
int totth = gridDim.x * blockDim.x;
int me = blockIdx.x*blockDim.x + threadIdx.x;
int i,j,k,sum = 0;

for(i = me; i<n; i+= totth){
for(j = i+i; j<n; j++){
for(k= 0; k<n ; k++)
       sum += m[n*i+k]*m[n*j+k];
}
}
atomicAdd(tot, sum);
}

int main(){
int n = 10, nblk = 4;
int *hm, *dm, htot, *dtot;

int msize = n*n*sizeof(int);

hm = (int *) malloc(msize);

int i,j;
for( i = 0; i < n; i++){
hm[n*i + i] = 0;
for(j=0;j<n; j++)
	     if(j != i) hm[i*n +j] = rand()%2;
}

hipMalloc((void **) &dm, msize);
hipMemcpy(dm, hm, msize, hipMemcpyHostToDevice);
htot = 0;
hipMalloc((void **) &dtot, sizeof(int));
hipMemcpy(dtot, &htot, sizeof(int), hipMemcpyHostToDevice);

dim3 dimGrid(nblk, 1);
dim3 dimBlock(192, 1, 1);

procpairs<<<dimGrid, dimBlock>>>(dm, dtot, n);

hipDeviceSynchronize();

hipMemcpy(&htot, dtot, sizeof(int), hipMemcpyDeviceToHost);

return 0;
}
