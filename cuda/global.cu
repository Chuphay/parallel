
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void kernel(int *array){

int index = blockIdx.x*blockDim.x + threadIdx.x;

array[index] = index;
}

int main(){
int num_elements = 256;
int num_bytes = num_elements*sizeof(int);

int *device_array = 0;
int *host_array = 0;

host_array = (int *)malloc(num_bytes);

hipMalloc((void **) &device_array, num_bytes);

int block_size = 128;
int grid_size = num_elements/block_size;

kernel<<<grid_size, block_size>>>(device_array);

hipMemcpy(host_array, device_array, num_bytes, hipMemcpyDeviceToHost);

int i;
for(i=0;i<num_elements; ++i)
printf("%d\n", host_array[i]);

free(host_array);
hipFree(device_array);

return 0;
} 