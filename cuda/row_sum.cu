#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void findleft(int *deviceMatrix, int *rowSum, int n){
int rownum = blockIdx.x;
int sum = 0;
int k;
for(k=0; k<n; k++)
	 sum += deviceMatrix[rownum*n+k];
rowSum[rownum] = sum;
}

int main(){
int n = 20;
int *hostMatrix, *deviceMatrix, *hostRowSum, *deviceRowSum;
int msize = n*n*sizeof(int);

hostMatrix = (int *) malloc(msize);
hostRowSum = (int *) malloc(n*sizeof(int));

hipMalloc((void **) &deviceMatrix, msize);
hipMalloc((void **) &deviceRowSum, msize/n);

int t = 0, i, j;
for(i = 0; i<n; i++){
for(j=0; j<n; j++)
	 hostMatrix[i*n+j] = t++;
}

hipMemcpy(deviceMatrix, hostMatrix, msize, hipMemcpyHostToDevice);

dim3 dimGrid(n,1);
dim3 dimBlock(1,1,1);

findleft<<<dimGrid, dimBlock>>>(deviceMatrix, deviceRowSum, n);

hipDeviceSynchronize();

hipMemcpy(hostRowSum, deviceRowSum, msize/n, hipMemcpyDeviceToHost);

for(i = 0; i<n; i++) printf("%d\n", hostRowSum[i]);
return 0;
}