
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

void square_this(int *array, int n){
  int i;
  for(i = 0 ; i<n ; i++)
    array[i] *= array[i];
}

__global__ void cuda_square_this(int *deviceArray){
  int me =threadIdx.x;
  
  deviceArray[me] *= 2;
}


int main(){

  int *hostArray, *deviceArray;

  int n = 10;
  int arraySize = n*sizeof(int);
  hostArray = (int *) malloc(arraySize);
  hipMalloc((void **) &deviceArray, arraySize);

  int i;
  for(i = 0; i<n ; i++)
    hostArray[i] = i+1;

  hipMemcpy(deviceArray, hostArray, arraySize, hipMemcpyHostToDevice);
  cuda_square_this<<<1,n>>>(deviceArray);


  square_this(hostArray, n);

  for(i = 0; i<n; i++)
    printf("%d\n",hostArray[i]);

  hipMemcpy(hostArray, deviceArray, arraySize, hipMemcpyDeviceToHost);

for(i=0;i<n;i++)
printf("%d\n", hostArray[i]);

  return 0;
}
