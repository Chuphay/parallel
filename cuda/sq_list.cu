#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

void square_this(int *array, int n){
  int i;
  for(i = 0 ; i<n ; i++)
    array[i] *= array[i];
}

__global__ cuda_square_this(int *deviceArray, int n){
  int i;
  for(i = 0; i<n; i++)
    deviceArray[i] *= deviceArray[i];
}


int main(){

  int *hostArray, *deviceArray;

  int n = 64;
  int arraySize = n*sizeof(int);
  hostArray = (int *) malloc(arraySize);
  hipMalloc((void **) &deviceArray, arraySize);

  int i;
  for(i = 0; i<n ; i++)
    hostArray[i] = i+1;

  square_this(hostArray, n);

  for(i = 0; i<n; i++)
    printf("%d\n",hostArray[i]);
  return 0;
}
