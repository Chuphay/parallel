#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void switch(int *deviceArray){

//extern __shared__ int sharedArray[];
//sharedArray[0] = deviceArray[1];
//deviceArray[2] = deviceArray[1];
//deviceArray[1] = sharedArray[0];
}

int main(){

int *hostArray, *deviceArray, n = 20;
int size = n*sizeof(int);

hostArray = (int *) malloc(size);
hipMalloc((void **) &deviceArray, size);
int i;
for(i = 0; i< n; i++)
      hostArray[i] = i;
hipMemcpy(deviceArray, hostArray, size, hipMemcpyHostToDevice);

dim3 dimGrid(1,1);
dim3 dimBlock(n,1,1);

switch <<<dimGrid, dimBlock, size>>> (deviceArray);

hipMemcpy(hostArray, deviceArray, size, hipMemcpyDeviceToHost);

for(i = 0; i<n; i++) printf("%d \n");

return 0;
}
